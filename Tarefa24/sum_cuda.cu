/*

1)

real    0m0,235s
user    0m0,059s
sys     0m0,155s

real    0m0,253s
user    0m0,067s
sys     0m0,167s

real    0m0,270s
user    0m0,075s
sys     0m0,166s

real    0m0,266s
user    0m0,091s
sys     0m0,152s

real    0m0,239s
user    0m0,073s
sys     0m0,143s

2)

==123303== NVPROF is profiling process 123303, command: ./a.out

Sum = 799999980000000.000000
==123303== Profiling application: ./a.out
==123303== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   82.35%  41.041ms         1  41.041ms  41.041ms  41.041ms  [CUDA memcpy HtoD]
                   17.60%  8.7735ms         1  8.7735ms  8.7735ms  8.7735ms  sum_cuda(double*, double*, int)

3)

real    0m0,068s
user    0m0,067s
sys     0m0,001s

real    0m0,062s
user    0m0,058s
sys     0m0,004s

real    0m0,043s
user    0m0,042s
sys     0m0,000s

real    0m0,082s
user    0m0,078s
sys     0m0,001s

real    0m0,063s
user    0m0,060s
sys     0m0,000s

4)

real    0m0,041s
user    0m0,187s
sys     0m0,004s

real    0m0,029s
user    0m0,152s
sys     0m0,000s

real    0m0,044s
user    0m0,139s
sys     0m0,004s

real    0m0,009s
user    0m0,048s
sys     0m0,004s

real    0m0,057s
user    0m0,296s
sys     0m0,004s

5)

real    0m0,032s
user    0m0,142s
sys     0m0,004s

real    0m0,065s
user    0m0,207s
sys     0m0,004s

real    0m0,042s
user    0m0,205s
sys     0m0,004s

real    0m0,058s
user    0m0,277s
sys     0m0,003s

real    0m0,070s
user    0m0,323s
sys     0m0,003s

6)

real    0m0,288s
user    0m0,101s
sys     0m0,163s

real    0m0,284s
user    0m0,089s
sys     0m0,171s

real    0m0,265s
user    0m0,090s
sys     0m0,154s

real    0m0,280s
user    0m0,074s
sys     0m0,186s

real    0m0,273s
user    0m0,106s
sys     0m0,146s

==125067== NVPROF is profiling process 125067, command: ./a.out

Sum = 799999980000000.000000
==125067== Profiling application: ./a.out
==125067== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   83.24%  49.618ms         1  49.618ms  49.618ms  49.618ms  [CUDA memcpy HtoD]
                   16.71%  9.9626ms         1  9.9626ms  9.9626ms  9.9626ms  sum_cuda(double*, double*, int)

Comentários:

Henrique, nós tentamos usar o servidor da PUC para pegar as métricas do CUDA com o comando nvprof, mas ele não listou tudo
que a gente precisava. Acreditamos que tem a ver com a versão do driver, porque testamos no PC de um dos integrantes do grupo
que tem uma GPU Nvidia com a versão mais nova do driver e funcionou. A GPU dele é uma Nvidia GTX 1060 6GB e a versão 
do driver é 470.74-10. Por causa disso, acreditamos que seria mais justo testar a versão OpenMP para GPU na mesma placa também.

A versão em CUDA rodou bem pior que as outras versões, e o grupo acredita que isso se deve à complexidade do código comparada 
à outras versões. Enquanto o código em CUDA precisa chamar uma função, rodar diversos loops for e fazer várias operações, as
outras versões são bem mais simples, tendo apenas 1 loop for.

Dentre as versões com e sem __shared__, podemos notar uma melhora considerável. A versão com uso do __shared__ tem uma melhora
enorme, pois o uso dessa flag faz uso da memória cache interna dos blocos da GPU, que é bem mais rápida que a memória global.

Entre as duas versões do código em CUDA, o maior overhead é causado entre a transferência dos dados entre a memória global da GPU
para as threads individuais. Como o uso da flag __shared__ favorece o uso da cache dos blocos de thread da GPU, esse overhead é
diminuído.

Obs: deixamos todos os códigos utilizados neste arquivo.

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void sum_cuda(double* a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;

  __shared__ double o[1024];  

  if(b+t < width)
    o[t] = a[b+t];

  __syncthreads();
  
  int i;
  for(i = blockDim.x/2; i > 0; i /= 2) {
    if(t < i && b+t+i < width)
      o[t] += o[t+i];
    
    __syncthreads();
  }

  if(t == 0)
    s[blockIdx.x] = o[0];  
} 

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // alocação e cópia dos dados
  hipMalloc((void **) &d_a, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  hipMalloc((void **) &d_s, s_size);

  // definição do número de blocos e threads
  dim3 dimGrid(num_blocks,1,1);
  dim3 dimBlock(block_size,1,1);

  // chamada do kernel
  sum_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);

  // cópia dos resultados para o host
  hipMemcpy(s, d_s, s_size, hipMemcpyDeviceToHost);

  // soma das reduções parciais
  for(int i = 1; i < num_blocks; i++) 
    s[0] += s[i];

  printf("\nSum = %f\n",s[0]);
  
  hipFree(d_a);
  hipFree(d_s);
}

/* sum-cuda-noshared
#include <stdio.h>
#include <stdlib.h>

__global__ void sum_cuda(double* a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;
 
  int i;
  for(i = blockDim.x/2; i > 0; i /= 2) {
    if(t < i && b+t+i < width)
      a[t+b] += a[t+b+i];
    
    __syncthreads();
  }
  if(t == 0)
    s[blockIdx.x] = a[t+b];  
} 

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // alocação e cópia dos dados
  cudaMalloc((void **) &d_a, size);
  cudaMemcpy(d_a, a, size, cudaMemcpyHostToDevice);

  cudaMalloc((void **) &d_s, s_size);

  // definição do número de blocos e threads
  dim3 dimGrid(num_blocks,1,1);
  dim3 dimBlock(block_size,1,1);

  // chamada do kernel
  sum_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);

  // cópia dos resultados para o host
  cudaMemcpy(s, d_s, s_size, cudaMemcpyDeviceToHost);

  // soma das reduções parciais
  for(int i = 1; i < num_blocks; i++) 
    s[0] += s[i];

  printf("\nSum = %f\n",s[0]);
  
  cudaFree(d_a);
  cudaFree(d_s);
}
*/

/* sum sequencial
#include <stdio.h>
#include <stdlib.h>

int main()
{
  int width = 40000000;
  double sum = 0;


  for(int i = 0; i < width; i++)
  {
     sum += i;
  }

  printf("\nSum = %f\n",sum);
}
*/

/* sum paralelo
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>


int main()
{
  double sum   =  0;
  int width    =  40000000;

  #pragma omp parallel for simd reduction(+:sum)
  for(int i = 0; i < width; i++)
  {
     sum += i;
  }

  printf("\nSum = %lf\n",sum);
}
*/

/* sum paralelo gpu
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>


int main()
{
  double sum   =  0;
  int width    =  40000000;

  #pragma omp target teams distribute parallel for simd map(tofrom:sum) map(to:width) reduction(+:sum) 
  for(int i = 0; i < width; i++)
  {
     sum += i;
  }

  printf("\nSum = %lf\n",sum);
}
*/
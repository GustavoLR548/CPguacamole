
/*

Tempo sequencial:

real    0m31,842s
user    0m31,713s
sys     0m0,083s

real    0m30,929s
user    0m30,789s
sys     0m0,074s

real    0m31,569s
user    0m31,408s
sys     0m0,077s

real    0m30,880s
user    0m30,749s
sys     0m0,067s

real    0m31,120s
user    0m30,967s
sys     0m0,087s

Tempo Paralelo - Multicore:

real    0m9,663s
user    1m1,498s
sys     0m2,830s

real    0m9,897s
user    1m1,072s
sys     0m2,809s

real    0m9,767s
user    1m0,906s
sys     0m2,798s

real    0m9,857s
user    1m1,442s
sys     0m2,972s

real    0m9,709s
user    1m1,580s
sys     0m3,002s

Tempo paralelo - GPU 

distribute 

real    0m30,728s
user    0m30,683s
sys     0m0,184s

real    0m30,883s
user    0m30,952s
sys     0m0,122s

real    0m31,138s
user    0m31,304s
sys     0m0,087s

real    0m31,675s
user    0m31,759s
sys     0m0,050s

real    0m31,450s
user    0m31,600s
sys     0m0,043s

distribute parallel for

real    0m9,370s
user    0m54,341s
sys     0m0,196s

real    0m9,421s
user    0m55,248s
sys     0m0,227s

real    0m8,649s
user    0m59,349s
sys     0m0,220s

real    0m8,653s
user    0m59,911s
sys     0m0,234s

real    0m8,644s
user    0m59,108s
sys     0m0,195s

distribute parallel for simd

real    0m9,144s
user    0m56,485s
sys     0m0,878s

real    0m9,119s
user    0m58,920s
sys     0m0,247s

real    0m8,835s
user    0m57,115s
sys     0m0,208s

real    0m8,842s
user    0m57,008s
sys     0m0,170s

real    0m8,984s
user    0m56,533s
sys     0m0,232s

Tempo paralelo do código em cuda:

real    0m0,803s
user    0m0,580s
sys     0m0,196s

real    0m0,789s
user    0m0,586s
sys     0m0,180s

real    0m0,720s
user    0m0,575s
sys     0m0,120s

real    0m0,693s
user    0m0,559s
sys     0m0,114s

real    0m0,752s
user    0m0,588s
sys     0m0,138s

Dados do nvprof

Invocations                                Event Name         Min         Max         Avg       Total
Device "GeForce GT 1030 (0)"
    Kernel: mm(double*, double*, double*, int)
          1                            warps_launched      277699      277699      277699      277699

==21368== Metric result:
Invocations                               Metric Name                        Metric Description         Min         Max         Avg
Device "GeForce GT 1030 (0)"
    Kernel: mm(double*, double*, double*, int)
          1                 warp_execution_efficiency                 Warp Execution Efficiency      99.21%      99.21%      99.21%


*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void mm(double* a, double* b, double* c, int width) 
{
	int j = blockIdx.y*blockDim.y+threadIdx.y;
	int i = blockIdx.x*blockDim.x+threadIdx.x;

	if((i < width) && (j < width))
	{	
		double sum = 0;
		for (int k = 0; k < width; k++) {
			double x = a[i * width + k];
			double y = b[k * width + j];
			sum += x * y;
		}
		c[i * width + j] = sum;
	}
}

int main()
{
	int width = 2000;
	double *a = (double*) malloc (width * width * sizeof(double));
	double *b = (double*) malloc (width * width * sizeof(double));
	double *c = (double*) malloc (width * width * sizeof(double));

	for(int i = 0; i < width; i++) {	  
		for(int j = 0; j < width; j++) {
			a[i*width+j] = i;
			b[i*width+j] = j;
			c[i*width+j] = 0;
		}
	}

	int size = width*width*sizeof(double);
	double *d_a, *d_b, *d_c;

	hipMalloc((void **) &d_a, size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_b, size);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_c, size);

	int block_size = 32;
	dim3 dimGrid((width-1)/block_size+1, (width-1)/block_size+1, 1);
	dim3 dimBlock(block_size, block_size, 1);

	mm<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, width);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	/*
	for(int i = 0; i < width; i++) {
		for(int j = 0; j < width; j++) {
			printf("\n c[%d][%d] = %lf",i,j,c[i*width+j]);
		}
	}
	*/

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
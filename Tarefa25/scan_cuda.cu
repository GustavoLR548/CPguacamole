/*

Tempo sequencial

real    0m0,178s
user    0m0,085s
sys     0m0,092s

real    0m0,164s
user    0m0,073s
sys     0m0,090s

real    0m0,163s
user    0m0,083s
sys     0m0,080s

real    0m0,167s
user    0m0,062s
sys     0m0,102s

real    0m0,166s
user    0m0,071s
sys     0m0,094s

Tempo paralelo

real    0m0,219s
user    0m0,030s
sys     0m0,169s

real    0m0,211s
user    0m0,027s
sys     0m0,163s

real    0m0,214s
user    0m0,027s
sys     0m0,167s

real    0m0,202s
user    0m0,023s
sys     0m0,160s

real    0m0,229s
user    0m0,036s
sys     0m0,182s

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void scan_cuda(double* a, double *s, int width) {
 int t = threadIdx.x;
 int b = blockIdx.x*blockDim.x; 
 double x;

 __shared__ double p[1024];

 if(b+t < width)
     p[t] = a[b+t];

 __syncthreads();

 for(int i = 1; i < blockDim.x; i *= 2) {
      if(t >= i)
          x = p[t] + p[t-i];

      __syncthreads();

      if(t >= i)
          p[t] = x;

      __syncthreads();
  }

 if(b + t < width)
     a[b+t] = p[t];

 if(t == blockDim.x-1)
     s[blockIdx.x+1] = a[b+t];

} 

__global__ void add_cuda(double *a, double *s, int width) {
  int t = threadIdx.x;                                                                                                           
  int b = blockIdx.x*blockDim.x; 
 
  if(b+t < width) 
      a[b+t] += s[blockIdx.x];
}




int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  s[0] = 0;
  for (int i = 1; i < num_blocks; i++)
    s[i] += s[i-1];

  printf("\na[%d] = %f\n",width-1,a[width-1]);
  
  hipFree(d_a);
  hipFree(d_s);
}